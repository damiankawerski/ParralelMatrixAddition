#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

const int size = 100000; 
const long long total_elements = static_cast<long long>(size) * size;
const int TILE_SIZE = 2000;  // Small tile size for better memory access patterns

// CUDA kernel do dodawania macierzy
__global__ void add_matrices(int* a, int* b, int* c, int tile_width, int tile_height, int full_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Simple bounds check
    if (idx < tile_width && idy < tile_height) {
        
        int index = idy * tile_width + idx;
        c[index] = a[index] + b[index];
    }
}

// Fill matrix with random values
void random_fill_matrix_tile(int* matrix, long long start_idx, long long elements) {
    for (long long i = 0; i < elements; ++i) {
        matrix[i] = rand() % 20001 - 10000;
    }
}

int main() {
    srand(time(nullptr));
    
    std::cout << "Starting matrix addition on RTX 4070 Ti (12GB VRAM)" << std::endl;
    std::cout << "Matrix size: " << size << "x" << size << " (" << (static_cast<double>(total_elements) * sizeof(int) / (1024.0 * 1024.0 * 1024.0)) << " GB per matrix)" << std::endl;
    std::cout << "Using tile size: " << TILE_SIZE << "x" << TILE_SIZE << std::endl;

    // For measuring kernel execution time
    double total_kernel_time = 0.0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Iterate over the matrix in tiles
    for (int y = 0; y < size; y += TILE_SIZE) {
        for (int x = 0; x < size; x += TILE_SIZE) {
            int current_tile_size_x = std::min(TILE_SIZE, size - x);
            int current_tile_size_y = std::min(TILE_SIZE, size - y);
            long long current_tile_elements = static_cast<long long>(current_tile_size_x) * current_tile_size_y;
            
            std::cout << "Processing tile at (" << x << "," << y << ") with size " 
                     << current_tile_size_x << "x" << current_tile_size_y << std::endl;

            // Allocate tiles on CPU
            int* h_tile1 = new int[current_tile_elements];
            int* h_tile2 = new int[current_tile_elements];
            int* h_result = new int[current_tile_elements];

            // Fill tiles with random values (outside of timing)
            random_fill_matrix_tile(h_tile1, 0, current_tile_elements);
            random_fill_matrix_tile(h_tile2, 0, current_tile_elements);

            // Allocate tiles on GPU
            int* d_tile1;
            int* d_tile2;
            int* d_result;
            hipMalloc(&d_tile1, current_tile_elements * sizeof(int));
            hipMalloc(&d_tile2, current_tile_elements * sizeof(int));
            hipMalloc(&d_result, current_tile_elements * sizeof(int));

            // Copy tiles from CPU to GPU (outside of timing)
            hipMemcpy(d_tile1, h_tile1, current_tile_elements * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_tile2, h_tile2, current_tile_elements * sizeof(int), hipMemcpyHostToDevice);

            // Kernel launch configuration
            dim3 threadsPerBlock(32, 32);
            dim3 numBlocks(
                (current_tile_size_x + threadsPerBlock.x - 1) / threadsPerBlock.x,
                (current_tile_size_y + threadsPerBlock.y - 1) / threadsPerBlock.y
            );

            // Start measuring time for the kernel
            hipEventRecord(start);

            // Run the kernel (addition)
            add_matrices<<<numBlocks, threadsPerBlock>>>(d_tile1, d_tile2, d_result, 
                                                       current_tile_size_x, current_tile_size_y, size);
            
            // Stop measuring time for the kernel
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            
            // Check for errors
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
                return -1;
            }

            // calculate elapsed time
            float kernel_milliseconds = 0;
            hipEventElapsedTime(&kernel_milliseconds, start, stop);
            total_kernel_time += kernel_milliseconds / 1000.0; // Konwersja z ms na sekundy
            
            // Copy result back to CPU (outside of timing)
            hipMemcpy(h_result, d_result, current_tile_elements * sizeof(int), hipMemcpyDeviceToHost);
            
            // Release GPU resources
            hipFree(d_tile1);
            hipFree(d_tile2);
            hipFree(d_result);
            
            // Release CPU resources
            delete[] h_tile1;
            delete[] h_tile2;
            delete[] h_result;
        }
    }

    // Clean up CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    std::cout << "Total kernel execution time: " << total_kernel_time << " seconds" << std::endl;
    std::cout << "Kernel throughput: " << (static_cast<double>(total_elements) * 2 * sizeof(int) / (1024.0 * 1024.0 * 1024.0)) / total_kernel_time 
              << " GB/s" << std::endl;

    return 0;
}

// Total kernel execution time: 2.23779 seconds
// Kernel throughput: 33.2944 GB/s